

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

#define MAX_THREADS_PER_BLOCK 1024

__global__ void findMaxInBlock(int *d_arr, int* gpu_return) {

	extern __shared__ int s_arr[];

	int tid = threadIdx.x;
	int index = blockIdx.x * MAX_THREADS_PER_BLOCK + threadIdx.x; 
	s_arr[tid] = d_arr[index];
	__syncthreads(); 

	for (int offset = 1; offset < blockDim.x; offset *=2) {

		if (tid % (2*offset) == 0) {
			if (s_arr[tid] < s_arr[tid + offset]) {
				s_arr[tid] = s_arr[tid + offset];
			}
		}
		__syncthreads();

	}

	if (tid == 0) {
		gpu_return[blockIdx.x] = s_arr[tid];
	}

}

void random_ints(int* a, int N) {

   int i;
   for (i = 0; i < N; ++i)
   		a[i] = rand() % (10000 + 1 - 0) + 0;

}

int main(void) {
	
	struct timeval cpu_start, cpu_end;
	struct timeval gpu_start, gpu_end;

	int *arr, *d_arr;
	int cpu_result, cpu_return; 
	int *gpu_result, *gpu_return;
	int N;
	scanf("%d", &N);

	int array_size = N * sizeof(int); 
	int integer_size = sizeof(int);

	int block_number = N / MAX_THREADS_PER_BLOCK;
	int thread_number = MAX_THREADS_PER_BLOCK;

	gpu_result = (int *)malloc(integer_size * block_number);

	arr = (int *)malloc(array_size); 
	random_ints(arr, N);

	hipMalloc((void **) &d_arr, array_size);

	hipMemcpy(d_arr, arr, array_size, hipMemcpyHostToDevice);

	gettimeofday(&gpu_start, NULL);

	findMaxInBlock<<<block_number
			,thread_number>>>
			(d_arr, gpu_return);

	gettimeofday(&gpu_end, NULL);

	hipMemcpy(gpu_result, gpu_return, integer_size * block_number, hipMemcpyDeviceToHost);
	
	cpu_return = 0;
	gettimeofday(&cpu_start, NULL);
	for (int i = 0; i < N; i++) {
		int num = arr[i];
		if (cpu_return < num) {
			cpu_return = num;
		}
	}
	gettimeofday(&cpu_end, NULL);
	cpu_result = cpu_return; 
	

	printf("GPU result: %d GPU run time: %ld\n", gpu_result[1],(gpu_end.tv_usec - gpu_start.tv_usec));
	printf("CPU result: %d CPU run time: %ld\n",cpu_result, (cpu_end.tv_usec - cpu_start.tv_usec));

}
