

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>


__global__ void sim_cell_onelife(int* d_cell, int m, int n) {

	 __shared__ int s_arr[100];
	 int neighbor = 0;  
	 
	 int r = threadIdx.x % m;
	 int c = threadIdx.x / m;
	 int cell = d_cell[r + c * m];
	 int row = r - 1; 
	 int col = c - 1; 


	 s_arr[r + c * m] = d_cell[r + c * m];
	 __syncthreads(); 

	 for (int i = row; i < row + 3; i++) {

	 	for (int j = col; j < col + 3; j++) {

	 		if (i >= 0 && j >= 0) {

	 			if (i < m && j < n) {

	 				if (i != r || j != c) {

	 					if (s_arr[i + j * m] == 1) {

	 						neighbor += 1; 

	 					}
	 				}
	 			}
	 		}
	 	}
	 }

	 if (cell == 1 && neighbor <= 1) {
	 	cell = 0; 
	 } else if (cell == 0 && neighbor == 2) {
	 	cell = 1;
	 } else if (cell == 0 && neighbor == 3){ 
	 	cell = 1; 
	 } else if (cell == 1 && neighbor >= 4) {
	 	cell = 0; 
	 }

	__syncthreads(); 

	d_cell[r + c * m] = cell;

}


int main(void) {
	
	int *cell; 
	int *d_cell; 
	int t_num; 
	int size; 
	int m, n, k; 

	struct timeval gpu_start, gpu_end;

	// m x n
	printf("Enter M: \n");
	scanf("%d", &m);

	printf("Enter N: \n");
	scanf("%d", &n);

	printf("Enter K: \n");
	scanf("%d", &k);

	t_num = n * m; 
	size = t_num * sizeof(int);

	cell = (int *)malloc(size);

	printf("-------BEFORE LIFE CYCLE--------\n");

	for (int i = 0; i < m; ++i) {
		for (int j = 0; j < n; ++j) {
			cell[i + j * m] = rand() % (1 - 0 + 1) + 0; 
		}
	}

	for (int i = 0; i < m; ++i) {
		for (int j = 0; j < n; ++j) {
			printf("%d ", cell[i + j * m]);
		}
		printf("\n");
	}		

	printf("-------AFTER %d LIFE CYCLES-------\n", k);
	hipMalloc((void **)&d_cell, size);
	
	hipMemcpy(d_cell, cell, size, hipMemcpyHostToDevice);

	gettimeofday(&gpu_start, NULL);
	for (int i = 0; i < k; i++) {
		sim_cell_onelife<<<1,t_num>>>(d_cell, m, n);
	}
	gettimeofday(&gpu_end, NULL);

	memset(cell, 0, size);

	hipMemcpy(cell, d_cell, size, hipMemcpyDeviceToHost);

	for (int i = 0; i < m; ++i) {
		for (int j = 0; j < n; ++j) {
			printf("%d ", cell[i + j * m]);
		}
		printf("\n");
	}	

	free(cell);
	hipFree(d_cell);
	printf("GPU run time: %ld\n",(gpu_end.tv_usec - gpu_start.tv_usec));

}